#include "hip/hip_runtime.h"
#include "../cueft.h"

// ================================
// SUM
// ================================

__global__ void
myblas_sum_x2_kernel_2 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	float *result
) {
}

__global__ void
myblas_sum_x2_kernel_2 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	double *result
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX2;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ double2 Ts2[2*2*MYGEMM_NTX2];
	double2 Tr2[2*2];

	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Tr2[iy*2+ix] = make_double2 (0., 0.);
		}
	}

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				Tr2[iy*2+ix] = cuQuadAdd (Tr2[iy*2+ix], w[ldw*(iy*2+ix)+i]);
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Ts2[nTx*(iy*2+ix)+iTx] = Tr2[iy*2+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 2; ix++) {
				#pragma unroll
				for (iy = 0; iy < 2; iy++) {
					Ts2[nTx*(iy*2+ix)+iTx] = cuQuadAdd (Ts2[nTx*(iy*2+ix)+iTx], Ts2[nTx*(iy*2+ix)+iTx+i]);
				}
			}
		}
		__syncthreads ();
	}
	if (iTx == 0 && iBx == 0) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				result[iy*2+ix] = Ts2[nTx*(iy*2+ix)].x;
				result[iy*2+ix+2*2] = Ts2[nTx*(iy*2+ix)].y;
			}
		}
	}
}

__global__ void
myblas_sum_x_kernel_2 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	float *result
) {
}

__global__ void
myblas_sum_x_kernel_2 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	double *result
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX2;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ double2 Ts2[2*2*MYGEMM_NTX2];
	double2 Tr2[2*2];

	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Tr2[iy*2+ix] = make_double2 (0., 0.);
		}
	}

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				Tr2[iy*2+ix] = cuQuadAdd (Tr2[iy*2+ix], w[ldw*(iy*2+ix)+i]);
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Ts2[nTx*(iy*2+ix)+iTx] = Tr2[iy*2+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 2; ix++) {
				#pragma unroll
				for (iy = 0; iy < 2; iy++) {
					Ts2[nTx*(iy*2+ix)+iTx] = cuQuadAdd (Ts2[nTx*(iy*2+ix)+iTx], Ts2[nTx*(iy*2+ix)+iTx+i]);
				}
			}
		}
		__syncthreads ();
	}
	if (iTx == 0) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				result[nBx*(iy*2+ix)+iBx] = Ts2[nTx*(iy*2+ix)].x + Ts2[nTx*(iy*2+ix)].y;
			}
		}
	}
}

template <typename TYPE>
__global__ void
myblas_sum_kernel_2 (
	const size_t n,
	const TYPE *w,
	const size_t ldw,
	TYPE *result
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX2;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ TYPE Ts1[2*2*MYGEMM_NTX2];
	TYPE Tr1[2*2] = {0.};

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				Tr1[iy*2+ix] += w[ldw*(iy*2+ix)+i];
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Ts1[nTx*(iy*2+ix)+iTx] = Tr1[iy*2+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 2; ix++) {
				#pragma unroll
				for (iy = 0; iy < 2; iy++) {
					Ts1[nTx*(iy*2+ix)+iTx] += Ts1[nTx*(iy*2+ix)+iTx+i];
				}
			}
		}
		__syncthreads ();
	}
	if (iTx == 0 && iBx == 0) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				result[iy*2+ix] = Ts1[nTx*(iy*2+ix)];
			}
		}
	}
}

// ================================
// DOT
// ================================

template <typename TYPE>
__global__ void
myblas_dot_kernel_2 (
	const size_t n,
	const TYPE * __restrict__ x,
	const size_t ldx,
	const TYPE * __restrict__ y,
	const size_t ldy,
	TYPE *w,
	const size_t ldw
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX2;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ TYPE Ts1[2*2*MYGEMM_NTX2];
	TYPE Xr1[2], Yr1[2];
	TYPE Tr1[2*2] = {0.};

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			Xr1[ix] = x[ix * ldx + i];
		}
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Yr1[iy] = y[iy * ldy + i];
		}
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				Tr1[iy*2+ix] += Xr1[ix] * Yr1[iy];
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Ts1[nTx*(iy*2+ix)+iTx] = Tr1[iy*2+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 2; ix++) {
				#pragma unroll
				for (iy = 0; iy < 2; iy++) {
					Ts1[nTx*(iy*2+ix)+iTx] += Ts1[nTx*(iy*2+ix)+iTx+i];
				}
			}
		}
		__syncthreads ();
	}

	if (iTx == 0) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				w[ldw*(iy*2+ix)+iBx] = Ts1[nTx*(iy*2+ix)];
			}
		}
	}
}

__global__ void
myblas_dot_x_kernel_2 (
	const size_t n,
	const float * __restrict__ x,
	const size_t ldx,
	const float * __restrict__ y,
	const size_t ldy,
	double2 *w,
	const size_t ldw
) {
}

__global__ void
myblas_dot_x_kernel_2 (
	const size_t n,
	const double * __restrict__ x,
	const size_t ldx,
	const double * __restrict__ y,
	const size_t ldy,
	double2 *w,
	const size_t ldw
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX2;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	register double Xr1[2], Yr1[2];
	register double2 Tr2[2*2];
	__shared__ double2 Ts2[2*2*MYGEMM_NTX2];

	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Tr2[iy*2+ix] = make_double2 (0., 0.);
		}
	}

	// DOT part -----------------------------------------------
	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) 
			Xr1[ix] = x[ix * ldx + i];
		#pragma unroll
		for (iy = 0; iy < 2; iy++) 
			Yr1[iy] = y[iy * ldy + i];
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				cuDot2i (Xr1[ix], Yr1[iy], Tr2[iy*2+ix]);
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 2; ix++) {
		#pragma unroll
		for (iy = 0; iy < 2; iy++) {
			Ts2[nTx*(iy*2+ix)+iTx] = Tr2[iy*2+ix];
		}
	}
	__syncthreads ();

	// SUM part -----------------------------------------------
	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 2; ix++) {
				#pragma unroll
				for (iy = 0; iy < 2; iy++) {
					Ts2[nTx*(iy*2+ix)+iTx] = cuQuadAdd (Ts2[nTx*(iy*2+ix)+iTx], Ts2[nTx*(iy*2+ix)+iTx+i]);
				}
			}
		}
		__syncthreads ();
	}

	if (iTx == 0) {
		#pragma unroll
		for (ix = 0; ix < 2; ix++) {
			#pragma unroll
			for (iy = 0; iy < 2; iy++) {
				w[ldw*(iy*2+ix)+iBx] = Ts2[nTx*(iy*2+ix)];
			}
		}
	}
}



