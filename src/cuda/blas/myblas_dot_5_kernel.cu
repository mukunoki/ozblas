#include "hip/hip_runtime.h"
#include "../cueft.h"

// ================================
// SUM
// ================================

__global__ void
myblas_sum_x2_kernel_5 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	float *result
) {
}

__global__ void
myblas_sum_x2_kernel_5 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	double *result
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX5;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ double2 Ts2[5*5*MYGEMM_NTX5];
	double2 Tr2[5*5];

	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Tr2[iy*5+ix] = make_double2 (0., 0.);
		}
	}

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				Tr2[iy*5+ix] = cuQuadAdd (Tr2[iy*5+ix], w[ldw*(iy*5+ix)+i]);
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Ts2[nTx*(iy*5+ix)+iTx] = Tr2[iy*5+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 5; ix++) {
				#pragma unroll
				for (iy = 0; iy < 5; iy++) {
					Ts2[nTx*(iy*5+ix)+iTx] = cuQuadAdd (Ts2[nTx*(iy*5+ix)+iTx], Ts2[nTx*(iy*5+ix)+iTx+i]);
				}
			}
		}
		__syncthreads ();
	}
	if (iTx == 0 && iBx == 0) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				result[iy*5+ix] = Ts2[nTx*(iy*5+ix)].x;
				result[iy*5+ix+5*5] = Ts2[nTx*(iy*5+ix)].y;
			}
		}
	}
}

__global__ void
myblas_sum_x_kernel_5 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	float *result
) {
}

__global__ void
myblas_sum_x_kernel_5 (
	const size_t n,
	const double2 * __restrict__ w,
	const size_t ldw,
	double *result
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX5;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ double2 Ts2[5*5*MYGEMM_NTX5];
	double2 Tr2[5*5];

	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Tr2[iy*5+ix] = make_double2 (0., 0.);
		}
	}

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				Tr2[iy*5+ix] = cuQuadAdd (Tr2[iy*5+ix], w[ldw*(iy*5+ix)+i]);
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Ts2[nTx*(iy*5+ix)+iTx] = Tr2[iy*5+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 5; ix++) {
				#pragma unroll
				for (iy = 0; iy < 5; iy++) {
					Ts2[nTx*(iy*5+ix)+iTx] = cuQuadAdd (Ts2[nTx*(iy*5+ix)+iTx], Ts2[nTx*(iy*5+ix)+iTx+i]);
				}
			}
		}
		__syncthreads ();
	}
	if (iTx == 0) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				result[nBx*(iy*5+ix)+iBx] = Ts2[nTx*(iy*5+ix)].x + Ts2[nTx*(iy*5+ix)].y;
			}
		}
	}
}

template <typename TYPE>
__global__ void
myblas_sum_kernel_5 (
	const size_t n,
	const TYPE *w,
	const size_t ldw,
	TYPE *result
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX5;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ TYPE Ts1[5*5*MYGEMM_NTX5];
	TYPE Tr1[5*5] = {0.};

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				Tr1[iy*5+ix] += w[ldw*(iy*5+ix)+i];
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Ts1[nTx*(iy*5+ix)+iTx] = Tr1[iy*5+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 5; ix++) {
				#pragma unroll
				for (iy = 0; iy < 5; iy++) {
					Ts1[nTx*(iy*5+ix)+iTx] += Ts1[nTx*(iy*5+ix)+iTx+i];
				}
			}
		}
		__syncthreads ();
	}
	if (iTx == 0 && iBx == 0) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				result[iy*5+ix] = Ts1[nTx*(iy*5+ix)];
			}
		}
	}
}

// ================================
// DOT
// ================================

template <typename TYPE>
__global__ void
myblas_dot_kernel_5 (
	const size_t n,
	const TYPE * __restrict__ x,
	const size_t ldx,
	const TYPE * __restrict__ y,
	const size_t ldy,
	TYPE *w,
	const size_t ldw
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX5;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	__shared__ TYPE Ts1[5*5*MYGEMM_NTX5];
	TYPE Xr1[5], Yr1[5];
	TYPE Tr1[5*5] = {0.};

	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			Xr1[ix] = x[ix * ldx + i];
		}
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Yr1[iy] = y[iy * ldy + i];
		}
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				Tr1[iy*5+ix] += Xr1[ix] * Yr1[iy];
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Ts1[nTx*(iy*5+ix)+iTx] = Tr1[iy*5+ix];
		}
	}
	__syncthreads ();

	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 5; ix++) {
				#pragma unroll
				for (iy = 0; iy < 5; iy++) {
					Ts1[nTx*(iy*5+ix)+iTx] += Ts1[nTx*(iy*5+ix)+iTx+i];
				}
			}
		}
		__syncthreads ();
	}

	if (iTx == 0) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				w[ldw*(iy*5+ix)+iBx] = Ts1[nTx*(iy*5+ix)];
			}
		}
	}
}

__global__ void
myblas_dot_x_kernel_5 (
	const size_t n,
	const float * __restrict__ x,
	const size_t ldx,
	const float * __restrict__ y,
	const size_t ldy,
	double2 *w,
	const size_t ldw
) {
}

__global__ void
myblas_dot_x_kernel_5 (
	const size_t n,
	const double * __restrict__ x,
	const size_t ldx,
	const double * __restrict__ y,
	const size_t ldy,
	double2 *w,
	const size_t ldw
) {
	const size_t iTx = threadIdx.x;
	const size_t iBx = blockIdx.x;
	const size_t nBx = gridDim.x;
	const size_t nTx = MYGEMM_NTX5;//blockDim.x;
	const size_t addrx = iBx * nTx + iTx;
	size_t i, ix, iy;
	register double Xr1[5], Yr1[5];
	register double2 Tr2[5*5];
	__shared__ double2 Ts2[5*5*MYGEMM_NTX5];

	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Tr2[iy*5+ix] = make_double2 (0., 0.);
		}
	}

	// DOT part -----------------------------------------------
	for (i = addrx; i < n; i += nTx * nBx) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) 
			Xr1[ix] = x[ix * ldx + i];
		#pragma unroll
		for (iy = 0; iy < 5; iy++) 
			Yr1[iy] = y[iy * ldy + i];
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				cuDot2i (Xr1[ix], Yr1[iy], Tr2[iy*5+ix]);
			}
		}
	}
	#pragma unroll
	for (ix = 0; ix < 5; ix++) {
		#pragma unroll
		for (iy = 0; iy < 5; iy++) {
			Ts2[nTx*(iy*5+ix)+iTx] = Tr2[iy*5+ix];
		}
	}
	__syncthreads ();

	// SUM part -----------------------------------------------
	#pragma unroll
	for (i = nTx/2; i > 0; i >>= 1) {
		if (iTx < i) {
			#pragma unroll
			for (ix = 0; ix < 5; ix++) {
				#pragma unroll
				for (iy = 0; iy < 5; iy++) {
					Ts2[nTx*(iy*5+ix)+iTx] = cuQuadAdd (Ts2[nTx*(iy*5+ix)+iTx], Ts2[nTx*(iy*5+ix)+iTx+i]);
				}
			}
		}
		__syncthreads ();
	}

	if (iTx == 0) {
		#pragma unroll
		for (ix = 0; ix < 5; ix++) {
			#pragma unroll
			for (iy = 0; iy < 5; iy++) {
				w[ldw*(iy*5+ix)+iBx] = Ts2[nTx*(iy*5+ix)];
			}
		}
	}
}



